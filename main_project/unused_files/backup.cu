#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <chrono>
#include <thread>


/* Device Number: 0
  Device name: NVIDIA GeForce RTX 4090
  Total global memory: 25756696576 bytes
  Shared memory per block: 49152 bytes
  Registers per block: 65536
  Warp size: 32
  Memory pitch: 2147483647 bytes
  Max threads per block: 1024
  Max threads dimensions: x = 1024, y = 1024, z = 64
  Max grid dimensions: x = 2147483647, y = 65535, z = 65535
  Clock rate: 2550000 kHz
  Total constant memory: 65536 bytes
  Compute capability: 8.9
  Texture alignment: 512 bytes
  Device overlap: 1
  Multi-processor count: 128
  Kernel execution timeout enabled: 1
  Integrated GPU sharing host memory: 0
  Can map host memory with hipHostAlloc/hipHostGetDevicePointer: 1
  Compute mode: 0
  Concurrent kernels: 1
  ECC enabled: 0
  PCI bus ID: 1
  PCI device ID: 0
  PCI domain ID: 0
  TCC driver mode: 0
  Async engine count: 1
  Unified addressing: 1
  Memory clock rate: 10501000 kHz
  Memory bus width: 384 bits
  L2 cache size: 75497472 bytes
  Max threads per multi-processor: 1536
  Stream priorities supported: 1
  Global L1 cache supported: 1
  Local L1 cache supported: 1
  Shared memory per multi-processor: 102400 bytes
  Registers per multi-processor: 65536
  Managed memory: 1
  Is multi-GPU board: 0
  Multi-GPU board group ID: 0
 */

// 总任务数量是: ALL_TASK_CNT = body_num*body_num/2 = 20224*10112 = 204,505,088 个计算任务
// 一共有128个SM
// 每个SM负载1,597,696个计算任务
// 每个Block最多1024个线程，从而平均每个线程1569.25个计算任务
// 从而设置TASK_BLOCK_SIZE = 40
// 一个任务块是: TASK_BLOCK_SIZE = *1024 = 1,048,576 个计算任务
// 总共需要的任务块数量: BLOCK_CNT = ALL_TASK_CNT/TASK_BLOCK_SIZE = 195
// 一个任务块执行所需内存加载量(Byte): 1024*2*size(float) = 8192 bytes
// 一个SM的总共享内存量: 102400 bytes

// 一个block的最大共享内存量: 49152 bytes
// 每个任务块对应一个线程，对应需要加载1024 Bytes到共享内存
// 
// block_num, thread_num是多少的时候可能整体效率最优？
//#define DATA_ID 0
#define G 1
#define TIME_TICK 0.016
#define SLOW_DOWN_FACTOR 0.001
#define BLOCK_SIZE 256 // 块的边长
#define EPSILON 1e-6
#define CHECK_IDX 10000
#define Y_LOW_BOUND 8192
#define Y_HIGH_BOUND 8447
#define detect_X 10000
#define detect_Y 5000
#define ADDITIONAL_ID 512
#define VERSION 5
#define SPEED_MAX 10
#define FLOAT_COMPENSATE_VALUE 0.001
// 0代表最初的版本（30ms），1代表最初版本的双精度版本（60ms），2代表大道至简法(4ms)，
// 3代表单线程法(非常多ms)，4代表第一次改进后的自定义方法(9ms)，5代表第二次改进后的自定义方法(2ms)

typedef float real;


float fov = 45.0f; // 视场角

bool isMousePressed = false;
double lastMouseX, lastMouseY;

float cameraDistance = 1.0;
glm::vec2 sphereCoords(0.0f, 0.0f); // 球坐标：theta和phi
glm::vec3 cameraPos = glm::vec3(0.0f, 0.0f, 1.0f);
glm::vec3 cameraTarget = glm::vec3(0.0f, 0.0f, 0.0f);
glm::vec3 cameraUp = glm::vec3(0.0f, 1.0f, 0.0f);
float  scaleFactors[3];

bool m_initialized;

// 速度buffer
real* m_deviceVelocity;

unsigned int m_numBodies;
unsigned int real_body_nums;

// 双缓冲处理，后来发现没必要删去了，因为渲染非常非常快
unsigned int m_pbo[2];
unsigned int m_currentRead;
unsigned int m_currentWrite;
hipGraphicsResource* m_pGRes[2]; // 与GL的缓冲区相绑定的




struct Header
{
    double time;
    int nbodies;
    int ndimension;
    int nsph;
    int ndark;
    int nstar;
};

struct DarkParticle
{
    real mass;
    real pos[3];
    real vel[3];
    real eps;
    int phi;
};

struct StarParticle
{
    real mass;
    real pos[3];
    real vel[3];
    real metals;
    real tform;
    real eps;
    int phi;
};

void initialize(int numBodies)
{
    m_numBodies = numBodies;
    unsigned int memSize = m_numBodies * 4 * sizeof(real);
    std::vector<real> position(m_numBodies * 4, 0);

    // generate double buffers.
    glGenBuffers(2, m_pbo);

    // initilize devices' buffers.
    glBindBuffer(GL_ARRAY_BUFFER, m_pbo[0]);
    glBufferData(GL_ARRAY_BUFFER, memSize, &position[0], GL_DYNAMIC_DRAW); // 初始化整个缓冲区
    int size = 0;
    glGetBufferParameteriv(GL_ARRAY_BUFFER, GL_BUFFER_SIZE, (GLint*)&size);
    if ((unsigned)size != memSize)
        fprintf(stderr, "WARNING: Pixel Buffer Object allocation failed!\n");
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    glBindBuffer(GL_ARRAY_BUFFER, m_pbo[1]);
    glBufferData(GL_ARRAY_BUFFER, memSize, &position[0], GL_DYNAMIC_DRAW); // 初始化整个缓冲区
    size = 0;
    glGetBufferParameteriv(GL_ARRAY_BUFFER, GL_BUFFER_SIZE, (GLint*)&size);
    if ((unsigned)size != memSize)
        fprintf(stderr, "WARNING: Pixel Buffer Object allocation failed!\n");
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // allocate velocity buffer.
    hipMalloc((void**)&m_deviceVelocity, memSize); // 为全体速度数据速度分配空间
    hipGraphicsGLRegisterBuffer(&m_pGRes[0], m_pbo[0], cudaGraphicsMapFlagsNone); // 将数字m_pbo[0]与缓冲区m_pGRes[0]绑定
    hipGraphicsGLRegisterBuffer(&m_pGRes[1], m_pbo[1], cudaGraphicsMapFlagsNone); // 将数字m_pbo[1]与缓冲区m_pGRes[1]绑定

    m_initialized = true;
}

void setParticlesPosition(real* data)
{
    // 初始化所有粒子的位置
    if (!m_initialized)
        return;
    m_currentRead = 0;
    m_currentWrite = 1;
    glBindBuffer(GL_ARRAY_BUFFER, m_pbo[m_currentRead]);
    glBufferSubData(GL_ARRAY_BUFFER, 0, 4 * sizeof(real) * m_numBodies, data); // 更新缓冲区的内容，从offset=0位置开始，更新数据量为所有天体的位置
    int size = 0;
    glGetBufferParameteriv(GL_ARRAY_BUFFER, GL_BUFFER_SIZE, (GLint*)&size); // 获取缓冲区大小
    if ((unsigned)size != 4 * (sizeof(real) * m_numBodies))
        printf( "WARNING: Pixel Buffer Object download failed!\n");
    glBindBuffer(GL_ARRAY_BUFFER, 0);

}

void setParticlesVelocity(real* data)
{
    // 初始化粒子速度
    if (!m_initialized)
        return;
    m_currentRead = 0;
    m_currentWrite = 1;

    hipMemcpy(m_deviceVelocity, data, m_numBodies * 4 * sizeof(real),
        hipMemcpyHostToDevice); //  m_deviceVelocity为当前的索引指针
}

void readTipsyFile(const std::string& fileName, std::vector<real>& positions, std::vector<real>& velocities, std::vector<int>& bodiesIds, int& nTotal, int& nFirst, int& nSecond, int& nThird)
{
    char fullFileName[256];
    sprintf(fullFileName, "%s", fileName.c_str());

    std::ifstream input(fullFileName, std::ios::in | std::ios::binary);
    if (!input.is_open())
    {
        std::cout << "Couldn't not open the tipsy file: " << fileName << std::endl;
        return;
    }

    Header header;
    input.read((char*)&header, sizeof(header));

    int idummy;
    glm::fvec4 pos;
    glm::fvec4 vel;

    nTotal = header.nbodies;
    real_body_nums = nTotal;
    nFirst = header.ndark;
    nSecond = header.nstar;
    nThird = header.nsph;
    positions.reserve(4 * nTotal);
    velocities.reserve(4 * nTotal);
    DarkParticle dark;
    StarParticle star;
    for (int index = 0; index < nTotal; ++index)
    {
        if (index < nFirst)
        {
            // dark particle.
            input.read((char*)&dark, sizeof(dark));
            vel.w = dark.eps;
            pos.w = dark.mass;
            pos.x = dark.pos[0];
            pos.y = dark.pos[1];
            pos.z = dark.pos[2];
            vel.x = dark.vel[0];
            vel.y = dark.vel[1];
            vel.z = dark.vel[2];
            idummy = dark.phi;
        }
        else
        {
            // star particle.
            input.read((char*)&star, sizeof(star));
            vel.w = star.eps;
            pos.w = star.mass;
            pos.x = star.pos[0];
            pos.y = star.pos[1];
            pos.z = star.pos[2];
            vel.x = star.vel[0];
            vel.y = star.vel[1];
            vel.z = star.vel[2];
            idummy = star.phi;
        }
        if (isnan(pos.x) || isnan(pos.y) || isnan(pos.z))
            std::cout << "Nan Error\n";
        positions.push_back(pos.x);
        positions.push_back(pos.y);
        positions.push_back(pos.z);
        positions.push_back(pos.w);
        velocities.push_back(vel.x);
        velocities.push_back(vel.y);
        velocities.push_back(vel.z);
        velocities.push_back(vel.w);
        bodiesIds.push_back(idummy);
    }

    // 填充到 BLOCK_SIZE 的倍数+1
    int newTotal = nTotal;
    if (nTotal % BLOCK_SIZE)
        newTotal = (nTotal / BLOCK_SIZE + 1) * BLOCK_SIZE;
    newTotal = newTotal + 1;//（由于实现的原因，需要总数量是BLOCK_SIZE + 1个）
    for (int index = nTotal; index < newTotal; ++index)
    {
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
        bodiesIds.push_back(index);
        ++nFirst;
    }
    nTotal = newTotal;


    input.close();
}

// 从文件读入天体数据，并加载到GPU上以及buffer上
// 该函数结束的时候，所有天体的位置数据都在m_pGRes[0]里面，初始速度则在m_deviceVelocity这个临时存储区里面。
void loadTipsyFile(const std::string& path)
{

    std::vector<int> ids;
    std::vector<real> positions;
    std::vector<real> velocities;

    int nBodies = 0;
    int nFirst = 0, nSecond = 0, nThird = 0;

    readTipsyFile(path, positions, velocities, ids,
        nBodies, nFirst, nSecond, nThird); 
    // 运行完毕后，positions中按照xyz weight的方式依次存入位置数据， velocities中按照xyz eps的方式依次存入速度数据，ids存入势能数据。
    // nFirst是dark的数量,nSecond是star的数量,nThird是SPH粒子的数量
    printf("nFirst=%d, nSecond=%d, nThird=%d\n", nFirst, nSecond, nThird);
    initialize(nBodies); // 按照数量初始化buffer
    setParticlesPosition(&positions[0]); // 传入的是全体数据的开头位置的指针, 将全体位置&重量数据保存到一个缓冲区里
    setParticlesVelocity(&velocities[0]); //传入的是全体数据的开头位置的指针，将全体速度数据写入到内存空间里
}

void readTabFile(
    const std::string& fileName,
    std::vector<real>& positions,
    std::vector<real>& velocities,
    int& nTotal)
{
    std::ifstream input;
    input.open(fileName, std::ifstream::in);
    if (input.fail())
    {
        std::cout << "Couldn't not open the tipsy file: " << fileName << std::endl;
        return;
    }
    std::string line;
    glm::vec4 pos, vel;
    nTotal = 0;
    bool first = true;
    bool second = true;
    while (!input.eof())
    {
        getline(input, line);
        std::istringstream iss(line.c_str());
        // mass x y z vx vy vz.
        iss >> pos.w >> pos.x >> pos.y >> pos.z >> vel.x >> vel.y >> vel.z;
        positions.push_back(pos.x);
        positions.push_back(pos.y);
        positions.push_back(pos.z);
        positions.push_back(pos.w);
        velocities.push_back(vel.x);
        velocities.push_back(vel.y);
        velocities.push_back(vel.z);
        velocities.push_back(vel.w);
        ++nTotal;
    }
    real_body_nums = nTotal;
    // round up to a multiple of 256 bodies.
    int newTotal = nTotal;
    if (nTotal % BLOCK_SIZE)
        newTotal = (nTotal / BLOCK_SIZE + 1) * BLOCK_SIZE;
    newTotal = newTotal + 1;
    for (int index = nTotal; index < newTotal; ++index)
    {
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
    }
    nTotal = newTotal;

    input.close();
}

void loadTabFile(const std::string& path)
{
    std::vector<real> positions;
    std::vector<real> velocities;

    int nBodies = 0;
    readTabFile(path, positions, velocities, nBodies);

    initialize(nBodies);
    setParticlesPosition(&positions[0]);
    setParticlesVelocity(&velocities[0]);
}

void readDatFile(
    const std::string& fileName,
    std::vector<real>& positions,
    std::vector<real>& velocities,
    int& nTotal)
{
    std::ifstream input;
    input.open(fileName, std::ifstream::in);
    if (input.fail())
    {
        std::cout << "Couldn't not open the tipsy file: " << fileName << std::endl;
        return;
    }
    std::string line;
    glm::vec4 pos, vel;
    nTotal = 0;
    pos.w = 1.0f;
    while (!input.eof())
    {
        getline(input, line);
        if (line.empty())
            continue;
        std::istringstream iss(line.c_str());
        // z y x vz vy vz.
        iss >> pos.z >> pos.y >> pos.x >> vel.z >> vel.y >> vel.x;
        positions.push_back(pos.x);
        positions.push_back(pos.y);
        positions.push_back(pos.z);
        positions.push_back(pos.w);
        velocities.push_back(vel.x);
        velocities.push_back(vel.y);
        velocities.push_back(vel.z);
        velocities.push_back(vel.w);
        ++nTotal;
    }
    real_body_nums = nTotal;
    // round up to a multiple of BLOCK_SIZE bodies.
    int newTotal = nTotal;
    if (nTotal % BLOCK_SIZE)
        newTotal = (nTotal / BLOCK_SIZE + 1) * BLOCK_SIZE;
    newTotal++;
    for (int index = nTotal; index < newTotal; ++index)
    {
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
    }
    nTotal = newTotal;

    input.close();
}

void loadDatFile(const std::string& path)
{
    // load the bodies from a dat file.
    std::vector<real> positions;
    std::vector<real> velocities;

    int nBodies = 0;
    readDatFile(path, positions, velocities, nBodies);

    initialize(nBodies);
    setParticlesPosition(&positions[0]);
    setParticlesVelocity(&velocities[0]);
}

void readSnapFile(
    const std::string& fileName,
    std::vector<real>& positions,
    std::vector<real>& velocities,
    int& nTotal)
{
    std::ifstream input;
    input.open(fileName, std::ifstream::in);
    if (input.fail())
    {
        std::cout << "Couldn't not open the tipsy file: " << fileName << std::endl;
        return;
    }
    std::string line;
    glm::vec4 pos, vel;
    nTotal = 0;

    // nbodies.
    getline(input, line);
    std::istringstream iss(line.c_str());
    iss >> nTotal;
    positions.resize(4 * nTotal);
    velocities.resize(4 * nTotal);

    // ndim.
    int dim = 0;
    getline(input, line);
    iss.str(line.c_str());
    iss >> dim;

    // time.
    float time;
    getline(input, line);
    iss.str(line.c_str());
    iss >> time;

    //masses.
    for (int x = 0; x < nTotal; ++x)
    {
        if (input.eof())
        {
            std::cout << "It's not a normal snap file:" << fileName << std::endl;
            return;
        }
        getline(input, line);
        std::istringstream iss(line.c_str());
        iss >> pos.w;
        positions[4 * x + 3] = pos.w;
    }

    // pos.
    for (int x = 0; x < nTotal; ++x)
    {
        if (input.eof())
        {
            std::cout << "It's not a normal snap file:" << fileName << std::endl;
            return;
        }
        getline(input, line);
        std::istringstream iss(line.c_str());
        iss >> pos.x >> pos.y >> pos.z;
        positions[4 * x + 0] = pos.x;
        positions[4 * x + 1] = pos.y;
        positions[4 * x + 2] = pos.z;
    }

    // vel.
    for (int x = 0; x < nTotal; ++x)
    {
        if (input.eof())
        {
            std::cout << "It's not a normal snap file:" << fileName << std::endl;
            return;
        }
        getline(input, line);
        std::istringstream iss(line.c_str());
        iss >> vel.x >> vel.y >> vel.z;
        velocities[4 * x + 0] = vel.x;
        velocities[4 * x + 1] = vel.y;
        velocities[4 * x + 2] = vel.z;
    }

    // eps.
    for (int x = 0; x < nTotal; ++x)
    {
        if (input.eof())
        {
            std::cout << "It's not a normal snap file:" << fileName << std::endl;
            return;
        }
        getline(input, line);
        std::istringstream iss(line.c_str());
        iss >> vel.w;
        velocities[4 * x + 3] = vel.w;
    }

    // round up to a multiple of BLOCK_SIZE bodies.
    real_body_nums = nTotal;
    int newTotal = nTotal;
    if (nTotal % BLOCK_SIZE)
        newTotal = (nTotal / BLOCK_SIZE + 1) * BLOCK_SIZE;
    newTotal++;
    for (int index = nTotal; index < newTotal; ++index)
    {
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        positions.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
        velocities.push_back(0.0f);
    }
    nTotal = newTotal;

    input.close();
}

void loadSnapFile(const std::string& path)
{
    std::vector<real> positions;
    std::vector<real> velocities;

    int nBodies = 0;
    readSnapFile(path, positions, velocities, nBodies);

    initialize(nBodies);
    setParticlesPosition(&positions[0]);
    setParticlesVelocity(&velocities[0]);
}

// 读取文件内容到字符串
std::string readFile(const char* filePath) {
    std::string content;
    std::ifstream fileStream(filePath, std::ios::in);

    if (!fileStream.is_open()) {
        std::cerr << "Could not read file " << filePath << ". File does not exist." << std::endl;
        return "";
    }

    std::string line = "";
    while (!fileStream.eof()) {
        std::getline(fileStream, line);
        content.append(line + "\n");
    }

    fileStream.close();
    return content;
}

// 检查着色器和程序的编译/链接错误
void checkShaderError(GLuint shader, GLuint flag, bool isProgram, const std::string& errorMessage) {
    GLint success = 0;
    GLchar error[1024] = { 0 };

    if (isProgram) {
        glGetProgramiv(shader, flag, &success);
    }
    else {
        glGetShaderiv(shader, flag, &success);
    }

    if (success == GL_FALSE) {
        if (isProgram)
            glGetProgramInfoLog(shader, sizeof(error), NULL, error);
        else
            glGetShaderInfoLog(shader, sizeof(error), NULL, error);

        std::cerr << errorMessage << ": '" << error << "'" << std::endl;
    }
}

// 加载并编译着色器，链接到程序中
GLuint loadShader(const char* vertex_path, const char* fragment_path) {
    std::string vertShaderStr = readFile(vertex_path);
    std::string fragShaderStr = readFile(fragment_path);
    const char* vertShaderSrc = vertShaderStr.c_str();
    const char* fragShaderSrc = fragShaderStr.c_str();

    GLuint vertShader = glCreateShader(GL_VERTEX_SHADER);
    GLuint fragShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(vertShader, 1, &vertShaderSrc, NULL);
    glShaderSource(fragShader, 1, &fragShaderSrc, NULL);

    glCompileShader(vertShader);
    checkShaderError(vertShader, GL_COMPILE_STATUS, false, "Vertex shader compilation failed");

    glCompileShader(fragShader);
    checkShaderError(fragShader, GL_COMPILE_STATUS, false, "Fragment shader compilation failed");

    GLuint shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertShader);
    glAttachShader(shaderProgram, fragShader);
    glLinkProgram(shaderProgram);
    checkShaderError(shaderProgram, GL_LINK_STATUS, true, "Shader program linking failed");

    glDeleteShader(vertShader);
    glDeleteShader(fragShader);

    return shaderProgram;
}

void processInput(GLFWwindow* window) {
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
}


/*  #region --------------------自己摸索的自以为的高级方法--------------------  */

// 计算出i和j之间的引力在引力数组中的index
__device__ int calculate_index(int N, int i, int j) {
    if (i > j) { // 交换i和j，使得j>=i
        int t = i;
        i = j;
        j = t;
    } 
    return i * (2 * N - i - 1) / 2 + j - i - 1;
}

// 使用block_id反推算group_id_x,group_id_y
// 
__device__ void calculate_group_x_y(int hori_group_nums, int block_id, int* group_id_x, int* group_id_y) {
    int x = 0;
    int y = 0;
    int line_len = hori_group_nums;
    int last = block_id + 1;  // 将block_id从0开始调整为从1开始

    while (last > 0) {
        last -= line_len;
        if (last > 0) {
            x++;
            line_len--;
        }
    }

    y = last + line_len + x - 1;  // 根据剩余的block_id计算y的值
    *group_id_x = x;
    *group_id_y = y;
}

// 从块间坐标(group_x, group_y)，转化为全矩阵坐标(x,y)
__device__ int calculate_matrix_index(int group_x, int group_y, int* x, int* y, int block_size) {
    *x = block_size * group_x;
    *y = block_size * group_y + 1;
}

// 计算单个引力
// x: 物体a的xyzw信息指针
// y: 物体b的xyzw信息指针
__device__ float3 cal_single_gravity(real* x, real* y) {
    // 数据组织：x,y,z,w（每个数据一个real类型大小）
    // F = G*m1*m2/r^2
    real dx = x[0] - y[0];
    real dy = x[1] - y[1];
    real dz = x[2] - y[2];
    real distance_squared = dx * dx + dy * dy + dz * dz;

    // 防止距离为零的情况，并避免除以零
    if (distance_squared < EPSILON) {
        distance_squared = EPSILON;
    }
    real distance = sqrt(distance_squared);
    real force =  G * x[3] * y[3] / (distance* distance* distance);
    float3 force_vector;
    force_vector.x = force * dx;
    force_vector.y = force * dy;
    force_vector.z = force * dz;
    return force_vector;
}

// 计算引力主函数
// 调用方式
// cal_gravity<<<block_num , block_edge_len, size>>>(gravity_array, (real*)position_and_weight, block_edge_len, hori_group_nums);
// size: 共享内存的大小，应为2*block_edge_len*4*sizeof(real)
// hori_group_nums: 一行中的最多总任务块数量
// block_edge_len: 任务块的边长
// gravity_array: 全局内存中的，引力数组，总长度为m(m-1)/2*size(real),计算结果要保存在这里
// position_and_weight: 全局内存中的位置质量数组,每个点占4*size(real)大小，总长度为m*4*size(real)
__global__ void cal_gravity(float3* gravity_array, real* position_and_weight, int block_edge_len, int hori_group_nums, int N) {
    extern __shared__ real mass[];  // 声明一个动态共享内存，保存该任务块要用到的质量&位置数组 （只保存各个列对应的物体，行对应的物体直接保存在线程内部）
    // size_t idx = blockIdx.x * blockDim.x + threadIdx.x;  // 计算当前线程应处理的天体索引
    int look_at = 16;
    // if (idx == look_at)printf("noew thread 0\n");
    // 任务块号是blockIdx.x
    // 线程在块内的编号是threadIdx.x
    // 块内线程总数为block_edge_len (= blockDim.x)
    // 根据任务块号和块内编号，加载数据到共享内存
    int group_id_x, group_id_y; // 块在块矩阵中的坐标
    int point_x, point_y, point_y_start; //块的左上角点在任务矩阵中的坐标
    calculate_group_x_y(hori_group_nums,blockIdx.x,&group_id_x, &group_id_y);
    //printf("idx = %d, block_id = %d, group_id_x = %d, group_id_y = %d\n", idx, blockIdx.x,group_id_x, group_id_y);
    calculate_matrix_index(group_id_x, group_id_y, &point_x, &point_y, block_edge_len);// 先找到在总matrix中的起始点，以定位当前处理的行
    //if (blockIdx.x % look_at == 0)printf("idx = %d, block_id = %d, group_id_x=%d, group_id_y=%d, point.x = %d, point.y = %d\n",idx, blockIdx.x, group_id_x, group_id_y, point_x, point_y);
    point_x = point_x + threadIdx.x; // 左上角坐标x->真实物体坐标x，同时也是本次计算的行号
    point_y_start = point_y;         // 本次计算的列起始号
    point_y = point_y + threadIdx.x; // 左上角坐标x->真实物体坐标y
    //// 未处理边缘区块数量不足的情况
    //if (point_x >= 20481) {
    //    printf("idx = %d,block_id = %d, group_id_x = %d, group_id_y = %d, x=%d, y=%d out of range\n", idx, blockIdx.x, group_id_x, group_id_y, point_y, point_y);
    //}
    //if (point_y >= 20481) {
    //    printf("idx = %d,block_id = %d, group_id_x = %d, group_id_y = %d, x=%d, y=%d out of range\n", idx, blockIdx.x, group_id_x, group_id_y, point_x, point_y);
    //}
    int share_pointer = threadIdx.x << 2;  // 等同于 threadIdx.x * 4
    int body_pointer_x = point_x << 2;     // 等同于 point_x * 4
    int body_pointer_y = point_y << 2;     // 等同于 point_y * 4
    real body_data[4] = {};     // 线程独享的数据
    for (int i = 0;i < 4;i++) { // 每个线程加载到共享内存一个点，block_edge_len个线程一共加载block_edge_len个点的数据
        body_data[i] = position_and_weight[body_pointer_x + i];
        mass[share_pointer + i] = position_and_weight[body_pointer_y + i];  
    }
    
    // 加载结束后,组内同步
    __syncthreads();
    
    // 正式开始重力计算
    // 计算任务是（point_x， point_y_start）一直到 （point_x， point_y_start+block_edge_len-1）
    // 也就是线程计算的是一行
    // 行对应的物体数据存在变量body_data里面，列对应的数据直接依次存在mass里面
    real* mass_pointer = mass;
    for (int i = 0;i < block_edge_len;i++) { // 边算边存，最大利用存储带宽
        float3 result = cal_single_gravity(body_data, mass_pointer);
        //if (i==0&&idx % 10000 == 0) {
        //    printf("idx = %zu, result = (%.4f, %.4f, %.4f)\n", result.x, result.y, result.z);
        //}
        int gravity_array_index = calculate_index(N, point_x, point_y_start + i);
        gravity_array[gravity_array_index] = result;
        mass_pointer += 4;
    }
}

// 将引力相加
// gravity_matrix_array，各个天体两两之间的引力矩阵，被拉平成一个数组，要结合calculate_index调用。
//  gravity_matrix_array[calculate_index(N, i, j)]就是引力矩阵中的第(i,j)位置的元素
// gravity_sum_array，结果目标数组，长度为N，存储其他天体对该天体的引力总和
// float3有三个分量 x,y,z，均为real类型
// 暂时是没有做进一步的并行。
__global__ void add_up_gravity(float3* gravity_matrix_array, float3* gravity_sum_array, int N) {
    // 由于计算index的时候i,j会自动处理，所以不用特别在意。但是不能使用(i,i)的值，因为对应的index是无效的。
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // 计算当前线程应处理的天体索引
    
    if (i < N) {
        float3 sum = { 0.0, 0.0, 0.0 };  // 初始化引力和为0

        for (int j = 0; j < N; j++) {
            if (i != j) {  // 避免计算天体对自身的引力
                int index = calculate_index(N, i, j);  // 获取拉平后数组中的索引
                float3 force = gravity_matrix_array[index];  // 从数组中取出引力

                sum.x += force.x;  // 累加x方向的引力
                sum.y += force.y;  // 累加y方向的引力
                sum.z += force.z;  // 累加z方向的引力
            }
        }

        gravity_sum_array[i] = sum;  // 将计算出的引力和存储到结果数组中
        //if (i % 10000 == 0) {
        //    //printf("i=%d force.x=%.4f\n", i, force.x);
        //    printf("i=%d gravity_sum  sum=（%.4f，%.4f ，%.4f ）\n", i, sum.x, sum.y, sum.z);
        //}
        //if (i == N - 2)printf("i=%d gravity_sum  sum=（%.4f，%.4f ，%.4f ） should be zero\n", i, sum.x, sum.y, sum.z);
    }
}

// 使用算好的引力直接更新全体物体速度
__global__ void update_position_and_speed(real* velocity_array, float3* gravity_sum_array, real* position_and_weight, int N) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;  // 计算当前线程应处理的天体索引
    if (i < N) { // 超过N的不再计算
        int base_index = i << 2;
        real mass = position_and_weight[base_index + 3];
        if (mass != 0.0f) {
            // mass是0的填充点不计算
            velocity_array[base_index] += gravity_sum_array[i].x * TIME_TICK / mass;
            velocity_array[base_index + 1] += gravity_sum_array[i].y * TIME_TICK / mass;
            velocity_array[base_index + 2] += gravity_sum_array[i].z * TIME_TICK / mass;

            position_and_weight[base_index] += velocity_array[base_index] * TIME_TICK;
            position_and_weight[base_index + 1] += velocity_array[base_index + 1] * TIME_TICK;
            position_and_weight[base_index + 2] += velocity_array[base_index + 2] * TIME_TICK;
        }
    }
}

__global__ void update_speed_half(real* velocity_array, float3* gravity_sum_array, real* position_and_weight, int N) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;  // 计算当前线程应处理的天体索引
    if (i < N) { // 超过N的不再计算
        int base_index = i << 2;
        real mass = position_and_weight[base_index + 3];
        if (mass != 0.0f) {
            // mass是0的填充点不计算
            velocity_array[base_index] += gravity_sum_array[i].x * TIME_TICK / mass / 2;
            velocity_array[base_index + 1] += gravity_sum_array[i].y * TIME_TICK / mass / 2;
            velocity_array[base_index + 2] += gravity_sum_array[i].z * TIME_TICK / mass / 2;
        }
    }
}

__global__ void update_position_complete(real * velocity_array, float3 * gravity_sum_array, real * position_and_weight, int N) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;  // 计算当前线程应处理的天体索引
    if (i < N) { // 超过N的不再计算
        int base_index = i << 2;
        real mass = position_and_weight[base_index + 3];
        if (mass != 0.0f) {
            // mass是0的填充点不计算
            position_and_weight[base_index] += velocity_array[base_index] * TIME_TICK;
            position_and_weight[base_index + 1] += velocity_array[base_index + 1] * TIME_TICK;
            position_and_weight[base_index + 2] += velocity_array[base_index + 2] * TIME_TICK;
        }
    }
}

/*  #endregion --------------------自定义的自以为的高级方法--------------------  */




/* #region --------------------反思之后找到的新的方法-------------------- */
__device__ float3 cal_single_acclerate_without_mass_new(real* a, real* b) {
    float3 dist;
    float3 acc;
    // a是主物体
    // b是作用力到a上的物体
    dist.x = b[0] - a[0];
    dist.y = b[1] - a[1];
    dist.z = b[2] - a[2];

    float distSquared = dist.x * dist.x + dist.y * dist.y + dist.z * dist.z;  // r^2
    distSquared += EPSILON; // r^2+epsilon
    float distance = sqrtf(distSquared); //  r+epsilon
    float coff = 1/(distance * distance * distance); //     compensate/r^3
    acc.x = dist.x*coff;
    acc.y = dist.y*coff;
    acc.z = dist.z*coff;
    return acc;
}

// 对float3类型进行atomic add
__device__ void atomicAdd3(float3* address, float3 val) {
    atomicAdd(&(address->x), val.x);
    atomicAdd(&(address->y), val.y);
    atomicAdd(&(address->z), val.z);
}

// 使用新的方法进行重力计算
// 每列中均进行加速度计算
// 
__global__ void cal_acc_new(float3* acc_array, real* position_and_weight, int block_edge_len, int hori_group_nums, int N) {
    extern __shared__ real sharedMemory[];  // 声明一个动态共享内存，前4*BlockSize个值保存该任务块要用到的质量&位置数组，后4*BlockSize个值保存纵向和 （只保存各个列对应的物体，行对应的物体直接保存在线程内部）
    real* mass = sharedMemory;
    real* column_acc_sum_list = sharedMemory + block_edge_len * 4;
    float3 local_acc_sum = { 0.0f,0.0f,0.0f };
    // size_t idx = blockIdx.x * blockDim.x + threadIdx.x;  // 计算当前线程应处理的天体索引
    int look_at = 16;
    // if (idx == look_at)printf("noew thread 0\n");
    // 任务块号是blockIdx.x
    // 线程在块内的编号是threadIdx.x
    // 块内线程总数为block_edge_len (= blockDim.x)
    // 根据任务块号和块内编号，加载数据到共享内存
    int group_id_x, group_id_y; // 块在块矩阵中的坐标
    int point_x, point_y, point_y_start; //块的左上角点在任务矩阵中的坐标
    calculate_group_x_y(hori_group_nums, blockIdx.x, &group_id_x, &group_id_y);
    //printf("idx = %d, block_id = %d, group_id_x = %d, group_id_y = %d\n", idx, blockIdx.x,group_id_x, group_id_y);
    calculate_matrix_index(group_id_x, group_id_y, &point_x, &point_y, block_edge_len);// 先找到在总matrix中的起始点，以定位当前处理的行
    //if (blockIdx.x % look_at == 0)printf("idx = %d, block_id = %d, group_id_x=%d, group_id_y=%d, point.x = %d, point.y = %d\n",idx, blockIdx.x, group_id_x, group_id_y, point_x, point_y);
    point_x = point_x + threadIdx.x; // 左上角坐标x->该线程负责计算的真实物体坐标x，同时也是本次计算的行号
    point_y_start = point_y;         // 本次计算的列起始号
    point_y = point_y + threadIdx.x; // 左上角坐标y->该线程负责加载和写入的物体坐标y

    int share_pointer = threadIdx.x*4;  // 等同于 threadIdx.x * 4
    int body_pointer_x = point_x*4;     // 等同于 point_x * 4
    int body_pointer_y = point_y*4;     // 等同于 point_y * 4
    real body_data[4] = {};     // 线程独享的数据
    for (int i = 0;i < 4;i++) { // 每个线程加载到共享内存一个点，block_edge_len个线程一共加载block_edge_len个点的数据
        body_data[i] = position_and_weight[body_pointer_x + i];
        mass[share_pointer + i] = position_and_weight[body_pointer_y + i];
    }
    // 加载结束后,组内同步
    __syncthreads();


    // 正式开始重力/加速度计算
    // 计算任务是（point_x， point_y_start）一直到 （point_x， point_y_start+block_edge_len-1）
    // 也就是线程计算的是一行
    // 行对应的物体数据存在变量body_data里面，列对应的数据直接依次存在mass里面
    int start_local_index = point_x - point_y_start + 1;
    if (start_local_index < 0) {
        start_local_index = 0;
    }

    real* mass_pointer = mass + 4 * start_local_index;
    real* column_sum_pointer = column_acc_sum_list + 4 * start_local_index;
    float3 single_acc;
    for (int i = start_local_index;i < block_edge_len;i++) { // 边算边存，最大利用存储带宽
        single_acc = cal_single_acclerate_without_mass_new(body_data, mass_pointer); // 算出（point_x, point_y+i）的重力值

        local_acc_sum.x += single_acc.x * mass_pointer[3];
        local_acc_sum.y += single_acc.y * mass_pointer[3];
        local_acc_sum.z += single_acc.z * mass_pointer[3];
        // 每列计算结束也不用同步,而是使用原子操作对共享内存进行写入
        // 此处是性能瓶颈
        atomicAdd(&column_sum_pointer[0], single_acc.x * body_data[3]);
        atomicAdd(&column_sum_pointer[1], single_acc.y * body_data[3]);
        atomicAdd(&column_sum_pointer[2], single_acc.z * body_data[3]);
        mass_pointer += 4;
        column_sum_pointer += 4;
    }

    __syncthreads();
    // 计算结束，同步一下，然后将结果保存到全局内存中，每个线程存两次即可。
    // 保存行和
    atomicAdd3(&acc_array[point_x], local_acc_sum);

    // 保存列和
    int column_sum_index = threadIdx.x * 4;
    single_acc.x = column_acc_sum_list[column_sum_index];
    single_acc.y = column_acc_sum_list[column_sum_index + 1];
    single_acc.z = column_acc_sum_list[column_sum_index + 2];
    atomicAdd3(&acc_array[point_y], single_acc);
}

// 在cal_acc_new的基础上的进一步改进
__global__ void cal_acc_advanced(float3* acc_array, real* position_and_weight, int block_edge_len, int hori_group_nums, int N) {
    extern __shared__ real sharedMemory[];  // 声明一个动态共享内存，前4*BlockSize个值保存该任务块要用到的质量&位置数组，后4*BlockSize个值保存纵向和 （只保存各个列对应的物体，行对应的物体直接保存在线程内部）
    real* mass = sharedMemory;
    real* column_acc_sum_list = sharedMemory + block_edge_len * 4;
    float3 local_acc_sum = { 0.0f,0.0f,0.0f };
    // size_t idx = blockIdx.x * blockDim.x + threadIdx.x;  // 计算当前线程应处理的天体索引
    int look_at = 16;
    // if (idx == look_at)printf("noew thread 0\n");
    // 任务块号是blockIdx.x
    // 线程在块内的编号是threadIdx.x
    // 块内线程总数为block_edge_len (= blockDim.x)
    // 根据任务块号和块内编号，加载数据到共享内存
    int group_id_x, group_id_y; // 块在块矩阵中的坐标
    int point_x, point_y, point_y_start; //块的左上角点在任务矩阵中的坐标
    calculate_group_x_y(hori_group_nums, blockIdx.x, &group_id_x, &group_id_y);
    //printf("idx = %d, block_id = %d, group_id_x = %d, group_id_y = %d\n", idx, blockIdx.x,group_id_x, group_id_y);
    calculate_matrix_index(group_id_x, group_id_y, &point_x, &point_y, block_edge_len);// 先找到在总matrix中的起始点，以定位当前处理的行
    //if (blockIdx.x % look_at == 0)printf("idx = %d, block_id = %d, group_id_x=%d, group_id_y=%d, point.x = %d, point.y = %d\n",idx, blockIdx.x, group_id_x, group_id_y, point_x, point_y);
    point_x = point_x + threadIdx.x; // 左上角坐标x->该线程负责计算的真实物体坐标x，同时也是本次计算的行号
    point_y_start = point_y;         // 本次计算的列起始号
    point_y = point_y + threadIdx.x; // 左上角坐标y->该线程负责加载和写入的物体坐标y

    int share_pointer = threadIdx.x * 4;  // 等同于 threadIdx.x * 4
    int body_pointer_x = point_x * 4;     // 等同于 point_x * 4
    int body_pointer_y = point_y * 4;     // 等同于 point_y * 4
    real body_data[4] = {};     // 线程独享的数据
    for (int i = 0;i < 4;i++) { // 每个线程加载到共享内存一个点，block_edge_len个线程一共加载block_edge_len个点的数据
        body_data[i] = position_and_weight[body_pointer_x + i];
        mass[share_pointer + i] = position_and_weight[body_pointer_y + i];
    }
    // 加载结束后,组内同步
    __syncthreads();


    // 正式开始重力/加速度计算
    // 计算任务是（point_x， point_y_start）一直到 （point_x， point_y_start+block_edge_len-1）
    // 也就是线程计算的是一行
    // 行对应的物体数据存在变量body_data里面，列对应的数据直接依次存在mass里面
    int is_diag;
    int task_len;
    if (point_x - point_y_start + 1 >= 0){
        is_diag = 1;
    }else {
        is_diag = 0;
    }
    int current_index = threadIdx.x; // 也就是offset
    if (is_diag == 1) {
        task_len = block_edge_len - current_index;
    }
    else {
        task_len = block_edge_len;
    }
    int max_index = block_edge_len;
    real* mass_pointer;
    real* column_sum_pointer;
    float3 single_acc;
    for (int i = 0;i < task_len; i++) { // 边算边存，最大利用存储带宽
        mass_pointer = mass + 4 * current_index;
        column_sum_pointer = column_acc_sum_list + 4 * current_index;
        int temp_current = point_y_start + current_index; // 应当是当前的y值
        single_acc = cal_single_acclerate_without_mass_new(body_data, mass_pointer); // 算出（point_x, point_y+i）的重力值
        //single_acc.x = single_acc.x / FLOAT_COMPENSATE_VALUE;
        //single_acc.y = single_acc.y / FLOAT_COMPENSATE_VALUE;
        //single_acc.y = single_acc.y / FLOAT_COMPENSATE_VALUE;
        //if (point_x == detect_X && temp_current == detect_Y || point_x == detect_Y && temp_current == detect_X) {
        //    printf("[%d]: (%.5f, %.5f, %.5f | %.5f)\n", point_x, body_data[0], body_data[1], body_data[2], body_data[3]);
        //    printf("[%d]: (%.5f, %.5f, %.5f | %.5f)\n", temp_current, mass_pointer[0], mass_pointer[1], mass_pointer[2], mass_pointer[3]);
        //    if (point_x == detect_Y && temp_current == detect_X) {
        //        printf("(%d,%d) acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", detect_X, detect_Y, -1*single_acc.x * body_data[3], -1*single_acc.y * body_data[3], -1*single_acc.z * body_data[3]);
        //    }else {
        //        printf("(%d,%d) acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", detect_X, detect_Y, single_acc.x * mass_pointer[3], single_acc.y * mass_pointer[3], single_acc.z * mass_pointer[3]);
        //    }
        //}
        //if (point_x== CHECK_IDX && temp_current >=Y_LOW_BOUND && temp_current <= Y_HIGH_BOUND) {
        //    printf("(%d,%d) acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", CHECK_IDX, point_y_start + current_index, single_acc.x * mass_pointer[3], single_acc.y * mass_pointer[3], single_acc.z * mass_pointer[3]);
        //}
        //if (temp_current == CHECK_IDX && point_x >= Y_LOW_BOUND && point_x <= Y_HIGH_BOUND) {
        //    printf("(%d,%d) acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", CHECK_IDX, point_x, -1*single_acc.x * body_data[3], -1*single_acc.y * body_data[3], -1*single_acc.z * body_data[3]);
        //}
        //if (-1 * single_acc.x * body_data[3]>40 || -1 * single_acc.y * body_data[3] > 40 || -1 * single_acc.z * body_data[3] > 40||- 1 * single_acc.x * body_data[3] <- 40 || -1 * single_acc.y * body_data[3] < -40 || -1 * single_acc.z * body_data[3] < -40) {
        //    printf("[%d]: (%.5f, %.5f, %.5f | %.5f)\n", point_x, body_data[0], body_data[1], body_data[2], body_data[3]);
        //    printf("[%d]: (%.5f, %.5f, %.5f | %.5f)\n", temp_current, mass_pointer[0], mass_pointer[1], mass_pointer[2], mass_pointer[3]);
        //    printf("(%d,%d) acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", CHECK_IDX, point_x, -1 * single_acc.x * body_data[3], -1 * single_acc.y * body_data[3], -1 * single_acc.z * body_data[3]);
        //}
        //if (-1 * single_acc.x * mass_pointer[3] > 40 || -1 * single_acc.y * mass_pointer[3] > 40 || -1 * single_acc.z * mass_pointer[3] > 40 || -1 * single_acc.x * mass_pointer[3] < -40 || -1 * single_acc.y * mass_pointer[3] < -40 || -1 * single_acc.z * mass_pointer[3] < -40) {
        //    printf("[%d]: (%.5f, %.5f, %.5f | %.5f)\n", point_x, body_data[0], body_data[1], body_data[2], body_data[3]);
        //    printf("[%d]: (%.5f, %.5f, %.5f | %.5f)\n", temp_current, mass_pointer[0], mass_pointer[1], mass_pointer[2], mass_pointer[3]);
        //    printf("(%d,%d) acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", CHECK_IDX, point_x, -1 * single_acc.x * body_data[3], -1 * single_acc.y * body_data[3], -1 * single_acc.z * body_data[3]);
        //}
        local_acc_sum.x += (single_acc.x*mass_pointer[3]);
        local_acc_sum.y += (single_acc.y*mass_pointer[3]);
        local_acc_sum.z += (single_acc.z*mass_pointer[3]);
        // 每列计算结束也不用同步,而是使用原子操作对共享内存进行写入
        // 此处是性能瓶颈,通过调整计算任务顺序，使得各个线程在同一时刻计算的任务不尽相同来减少共享内存冲突。
        atomicAdd(&column_sum_pointer[0], single_acc.x * body_data[3]);
        real middle_result = atomicAdd(&column_sum_pointer[1], single_acc.y * body_data[3]);
        //if (temp_current == CHECK_IDX && point_x <= 256) {
        //    printf("%d: [%d]= %.5f+%.5f -> %.5f\n", point_x, middle_result, single_acc.y * body_data[3], column_sum_pointer[1]);
        //}
        atomicAdd(&column_sum_pointer[2], single_acc.z * body_data[3]);
        
        current_index = (current_index + 1) % max_index;
    }

    __syncthreads();
    // 计算结束，同步一下，然后将结果保存到全局内存中，每个线程存两次即可。
    //// 保存行和
    //if (point_x == CHECK_IDX && group_id_x == CHECK_IDX/BLOCK_SIZE) {
    //    int start_gid = group_id_y * BLOCK_SIZE+1;
    //    int end_gid = start_gid + BLOCK_SIZE-1;
    //    printf("%d: %d->%d sum=(%.4f, %.4f, %.4f)\n", CHECK_IDX, start_gid, end_gid, local_acc_sum.x, local_acc_sum.y, local_acc_sum.z);
    //}
    atomicAdd3(&acc_array[point_x], local_acc_sum);

    // 保存列和
    int column_sum_index = threadIdx.x * 4;
    single_acc.x = -1*column_acc_sum_list[column_sum_index]; // 因为主物体反过来了，所以加速度要反过来
    single_acc.y = -1*column_acc_sum_list[column_sum_index + 1];
    single_acc.z = -1*column_acc_sum_list[column_sum_index + 2];
    //if (point_y == CHECK_IDX) {
    //    printf("%d: %d->%d sum=(%.4f, %.4f, %.4f) [column] \n", point_y ,(group_id_x*BLOCK_SIZE), (group_id_x * BLOCK_SIZE+ BLOCK_SIZE-1),single_acc.x, single_acc.y, single_acc.z);
    //}
    atomicAdd3(&acc_array[point_y], single_acc);
}

// 使用计算好的加速度，以及全局内存中的速度和原始位置信息来更新物体位置
__global__ void use_acc_update_position(real* position_and_weight, real* m_deviceVelocity, float3* acc_array) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;  // 计算当前线程应处理的天体索引
    float3 accelerate = acc_array[idx];
    if (idx == CHECK_IDX) {
        printf("%d: acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", CHECK_IDX, accelerate.x, accelerate.y, accelerate.z);
    }
    acc_array[idx] = { 0.0f, 0.0f, 0.0f }; // 用完清空
    // 使用局部变量来减少对全局内存的访问
    real vx = m_deviceVelocity[idx * 4];
    real vy = m_deviceVelocity[idx * 4 + 1];
    real vz = m_deviceVelocity[idx * 4 + 2];

    //if (vx >= SPEED_MAX || vy >= SPEED_MAX || vz >= SPEED_MAX) {
    //    printf("%d overspeed! speed=(%.5f, %.5f, %.5f)\n", idx, vx, vy, vz);
    //}

    vx += accelerate.x * TIME_TICK;
    vy += accelerate.y * TIME_TICK;
    vz += accelerate.z * TIME_TICK;
    if (idx == CHECK_IDX) {
        printf("%d: speed.x=%.4f, speed.y=%.4f, speed.z=%.4f\n", CHECK_IDX, vx, vy, vz);
    }

    // 将更新后的速度写回全局内存
    m_deviceVelocity[idx * 4] = vx;
    m_deviceVelocity[idx * 4 + 1] = vy;
    m_deviceVelocity[idx * 4 + 2] = vz;

    // 更新位置
    position_and_weight[idx * 4] += vx * TIME_TICK;
    position_and_weight[idx * 4 + 1] += vy * TIME_TICK;
    position_and_weight[idx * 4 + 2] += vz * TIME_TICK;

    if (idx == CHECK_IDX) {
        printf("%d: pos.x=%.4f, pos.y=%.4f, pos.z=%.4f\n", CHECK_IDX, position_and_weight[idx * 4], position_and_weight[idx * 4 + 1], position_and_weight[idx * 4 + 2]);
    }

}

/* #endregion --------------------反思后优化的新方法-------------------- */





/*  #region --------------------简单，但确实有效的方法--------------------  */

__device__ float3 cal_single_acclerate(real* a, real* b, float3 acc) {
    float3 dist;
    // a是主物体
    // b是作用力到a上的物体
    dist.x = b[0] - a[0];
    dist.y = b[1] - a[1];
    dist.z = b[2] - a[2];
    
    float distSquared = dist.x * dist.x + dist.y * dist.y + dist.z * dist.z;
    distSquared += EPSILON;
    float distance = sqrtf(distSquared);
    float coff = b[3] / (distance* distance* distance);
    acc.x += dist.x * coff;
    acc.y += dist.y * coff;
    acc.z += dist.z * coff;
    return acc;
}

// 调用线程数量 = 物体的数量
// 线程块大小还是1024
__global__ void simple_update_all( real* m_deviceVelocity, real* position_and_weight, int N) {
    extern __shared__ real mass[];
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;  // 计算当前线程应处理的天体索引
    if (idx < N) {
        real main_body[4];
        main_body[0] = position_and_weight[idx*4];
        main_body[1] = position_and_weight[idx*4 + 1];
        main_body[2] = position_and_weight[idx*4 + 2];
        main_body[3] = position_and_weight[idx*4 + 3];
        float3 accelerate = {0.0f, 0.0f, 0.0f};
        size_t  now_pos = 0;
        int k = threadIdx.x;
        float3 last_acc = { 0.0f, 0.0f, 0.0f };
        while (now_pos < 4 * N) {
            // 一个线程加载一个数据，然后计算BLOCK_SIZE个，然后同步一下，然后再加载数据到共享内存，然后再同步一下
            mass[k*4] = position_and_weight[now_pos+ k*4];
            mass[k*4 + 1] = position_and_weight[now_pos + k*4 + 1];
            mass[k*4 + 2] = position_and_weight[now_pos + k*4 + 2];
            mass[k*4 + 3] = position_and_weight[now_pos + k*4 + 3];
            
            __syncthreads(); // 确保大家都加载完成

            for (int i = 0;i < BLOCK_SIZE;i++) {
                int current = i + now_pos / 4;
                if (idx == detect_X && current ==detect_Y) {
                    float3 acc_before = accelerate;
                    //printf("[%d]: (%.5f, %.5f, %.5f | %.5f)\n", idx, main_body[0], main_body[1], main_body[2], main_body[3]);
                    //printf("[%d]: (%.5f, %.5f, %.5f | %.5f)\n", current, mass[i * 4], mass[i * 4 + 1], mass[i * 4 + 2], mass[i * 4 + 3]);
                    //accelerate = cal_single_acclerate(main_body, &mass[i * 4], accelerate);
                    //printf("(%d,%d) acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", CHECK_IDX, current, accelerate.x - acc_before.x, accelerate.y - acc_before.y, accelerate.z - acc_before.z);
                }else if (idx == CHECK_IDX && ((current >= Y_LOW_BOUND && current <= Y_HIGH_BOUND) || current % BLOCK_SIZE == 0 )) { //|| current%BLOCK_SIZE==0
                    float3 acc_before = accelerate;
                    /*accelerate = cal_single_acclerate(main_body, &mass[i * 4], accelerate);
                    printf("(%d,%d) acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", CHECK_IDX, current, accelerate.x - acc_before.x, accelerate.y - acc_before.y, accelerate.z - acc_before.z);*/
                }else{
                    float3 acc_before = accelerate;
                    accelerate = cal_single_acclerate(main_body, &mass[i * 4], accelerate);
                    float3 new_acc = { accelerate.x - acc_before.x, accelerate.y - acc_before.y, accelerate.z - acc_before.z };
                    if (new_acc.x > 40 || new_acc.y > 40 || new_acc.z > 40 || new_acc.x < -40 || new_acc.y < -40 || new_acc.z < -40) {
                        printf("[%d]: (%.5f, %.5f, %.5f | %.5f)\n", idx, main_body[0], main_body[1], main_body[2], main_body[3]);
                        printf("[%d]: (%.5f, %.5f, %.5f | %.5f)\n", current, mass[i * 4], mass[i * 4 + 1], mass[i * 4 + 2], mass[i * 4 + 3]);
                        printf("(%d,%d) acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", CHECK_IDX, current, new_acc.x, new_acc.y, new_acc.z);
                    }
                }
            }
            
            __syncthreads(); // 确保大家都计算完毕
            //if (idx == CHECK_IDX) {
            //    int temp_last_idx = now_pos / 4;
            //    int temp_current_idx = now_pos / 4 + BLOCK_SIZE-1;
            //    printf(" %d: %d->%d sum=(%.4f, %.4f, %.4f)\n", CHECK_IDX, temp_last_idx, temp_current_idx, accelerate.x-last_acc.x, accelerate.y - last_acc.y, accelerate.z - last_acc.z);
            //    last_acc = accelerate;
            //}
            now_pos += BLOCK_SIZE * 4;
        }
        // printf("now_pos_final=%d\n", now_pos);

        // 到这里，一个天体的加速度已经计算完成下，然后我们计算更新他的速度

        // float abcd = accelerate.y;

        //m_deviceVelocity[idx * 4] += accelerate.x * TIME_TICK;
        //m_deviceVelocity[idx*4+1] += accelerate.y * TIME_TICK;
        //m_deviceVelocity[idx*4+2] += accelerate.z * TIME_TICK;

        //// 最后更新他的位置
        //position_and_weight[idx*4] += m_deviceVelocity[idx*4] * TIME_TICK;
        //position_and_weight[idx*4+1] += m_deviceVelocity[idx*4+1] * TIME_TICK;
        //position_and_weight[idx*4+2] += m_deviceVelocity[idx*4+2] * TIME_TICK;
        if (idx == CHECK_IDX) {
            printf("%d: acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", CHECK_IDX,  accelerate.x, accelerate.y, accelerate.z);
        }

        // 加速器和时间步长
        float ax = accelerate.x * TIME_TICK;
        float ay = accelerate.y * TIME_TICK;
        float az = accelerate.z * TIME_TICK;

        // 更新速度
        m_deviceVelocity[idx * 4] += ax;
        m_deviceVelocity[idx * 4 + 1] += ay;
        m_deviceVelocity[idx * 4 + 2] += az;

        // 将更新后的速度存入局部变量
        float vx = m_deviceVelocity[idx * 4];
        float vy = m_deviceVelocity[idx * 4 + 1];
        float vz = m_deviceVelocity[idx * 4 + 2];

        //if (vx >= SPEED_MAX || vy >= SPEED_MAX || vz >= SPEED_MAX) {
        //    printf("%d overspeed! speed=(%.5f, %.5f, %.5f)\n", idx, vx, vy, vz);
        //}

        if (idx == CHECK_IDX) {
            printf("%d: speed.x=%.4f, speed.y=%.4f, speed.z=%.4f\n", CHECK_IDX,  vx, vy, vz);
        }

        // 更新位置
        position_and_weight[idx * 4] += vx * TIME_TICK;
        position_and_weight[idx * 4 + 1] += vy * TIME_TICK;
        position_and_weight[idx * 4 + 2] += vz * TIME_TICK;

        if (idx == CHECK_IDX) {
            printf("%d: pos.x=%.4f, pos.y=%.4f, pos.z=%.4f\n", CHECK_IDX, position_and_weight[idx * 4], position_and_weight[idx * 4 + 1], position_and_weight[idx * 4 + 2]);
        }
    }
}

//// 调用线程数量 = 物体的数量
//// 线程块大小还是1024
//__global__ void simple_update_all(real* m_deviceVelocity, real* position_and_weight, int N) {
//    extern __shared__ real mass[];
//    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;  // 计算当前线程应处理的天体索引
//    if (idx < N) {
//        real main_body[4];
//        main_body[0] = position_and_weight[idx * 4];
//        main_body[1] = position_and_weight[idx * 4 + 1];
//        main_body[2] = position_and_weight[idx * 4 + 2];
//        main_body[3] = position_and_weight[idx * 4 + 3];
//        float3 accelerate = { 0.0f, 0.0f, 0.0f };
//        size_t  now_pos = 0;
//        int k = threadIdx.x;
//
//        while (now_pos < 4 * N) {
//            // 一个线程加载一个数据，然后计算BLOCK_SIZE个，然后同步一下，然后再加载数据到共享内存，然后再同步一下
//            mass[k * 4] = position_and_weight[now_pos + k * 4];
//            mass[k * 4 + 1] = position_and_weight[now_pos + k * 4 + 1];
//            mass[k * 4 + 2] = position_and_weight[now_pos + k * 4 + 2];
//            mass[k * 4 + 3] = position_and_weight[now_pos + k * 4 + 3];
//
//            __syncthreads(); // 确保大家都加载完成
//
//            for (int i = 0;i < BLOCK_SIZE;i++) {
//                if (idx == CHECK_IDX && now_pos == 0 && i < 10) {
//                    float3 acc_before = accelerate;
//                    accelerate = cal_single_acclerate(main_body, &mass[i * 4], accelerate);
//                    printf("(%d,%d) acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", CHECK_IDX, now_pos + i, accelerate.x - acc_before.x, accelerate.y - acc_before.y, accelerate.z - acc_before.z);
//                }
//                else {
//                    accelerate = cal_single_acclerate(main_body, &mass[i * 4], accelerate);
//                }
//            }
//
//            __syncthreads(); // 确保大家都计算完毕
//            now_pos += BLOCK_SIZE * 4;
//        }
//        // printf("now_pos_final=%d\n", now_pos);
//
//        // 到这里，一个天体的加速度已经计算完成下，然后我们计算更新他的速度
//
//        // float abcd = accelerate.y;
//
//        //m_deviceVelocity[idx * 4] += accelerate.x * TIME_TICK;
//        //m_deviceVelocity[idx*4+1] += accelerate.y * TIME_TICK;
//        //m_deviceVelocity[idx*4+2] += accelerate.z * TIME_TICK;
//
//        //// 最后更新他的位置
//        //position_and_weight[idx*4] += m_deviceVelocity[idx*4] * TIME_TICK;
//        //position_and_weight[idx*4+1] += m_deviceVelocity[idx*4+1] * TIME_TICK;
//        //position_and_weight[idx*4+2] += m_deviceVelocity[idx*4+2] * TIME_TICK;
//        if (idx == CHECK_IDX) {
//            printf("%d: acc.x=%.4f, acc.y=%.4f, acc.z=%.4f\n", CHECK_IDX, accelerate.x, accelerate.y, accelerate.z);
//        }
//
//        // 加速器和时间步长
//        float ax = accelerate.x * TIME_TICK;
//        float ay = accelerate.y * TIME_TICK;
//        float az = accelerate.z * TIME_TICK;
//
//        // 更新速度
//        m_deviceVelocity[idx * 4] += ax;
//        m_deviceVelocity[idx * 4 + 1] += ay;
//        m_deviceVelocity[idx * 4 + 2] += az;
//
//        // 将更新后的速度存入局部变量
//        float vx = m_deviceVelocity[idx * 4];
//        float vy = m_deviceVelocity[idx * 4 + 1];
//        float vz = m_deviceVelocity[idx * 4 + 2];
//
//        if (idx == CHECK_IDX) {
//            printf("%d: speed.x=%.4f, speed.y=%.4f, speed.z=%.4f\n", CHECK_IDX, vx, vy, vz);
//        }
//
//        // 更新位置
//        position_and_weight[idx * 4] += vx * TIME_TICK;
//        position_and_weight[idx * 4 + 1] += vy * TIME_TICK;
//        position_and_weight[idx * 4 + 2] += vz * TIME_TICK;
//
//        if (idx == CHECK_IDX) {
//            printf("%d: pos.x=%.4f, pos.y=%.4f, pos.z=%.4f\n", CHECK_IDX, position_and_weight[idx * 4], position_and_weight[idx * 4 + 1], position_and_weight[idx * 4 + 2]);
//        }
//    }
//}

// 更高精度的update
//__global__ void simple_update_all(real* m_deviceVelocity, real* position_and_weight, int N) {
//    extern __shared__ real mass[];
//    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if (idx < N) {
//        real main_body[4];
//        main_body[0] = position_and_weight[idx * 4];
//        main_body[1] = position_and_weight[idx * 4 + 1];
//        main_body[2] = position_and_weight[idx * 4 + 2];
//        main_body[3] = position_and_weight[idx * 4 + 3];
//        float3 accelerate = { 0.0f, 0.0f, 0.0f };
//        size_t now_pos = 0;
//        int k = threadIdx.x;
//
//        // First half-step velocity update
//        while (now_pos < 4 * N) {
//            mass[k * 4] = position_and_weight[now_pos + k * 4];
//            mass[k * 4 + 1] = position_and_weight[now_pos + k * 4 + 1];
//            mass[k * 4 + 2] = position_and_weight[now_pos + k * 4 + 2];
//            mass[k * 4 + 3] = position_and_weight[now_pos + k * 4 + 3];
//
//            now_pos += BLOCK_SIZE * 4;
//            __syncthreads();
//
//            for (int i = 0; i < BLOCK_SIZE; i++) {
//                accelerate = cal_single_acclerate(main_body, &mass[i * 4], accelerate);
//            }
//            __syncthreads();
//        }
//
//        // Update velocity with first half-step acceleration
//        m_deviceVelocity[idx * 4] += accelerate.x * (TIME_TICK / 2.0f);
//        m_deviceVelocity[idx * 4 + 1] += accelerate.y * (TIME_TICK / 2.0f);
//        m_deviceVelocity[idx * 4 + 2] += accelerate.z * (TIME_TICK / 2.0f);
//
//        // Update position
//        position_and_weight[idx * 4] += m_deviceVelocity[idx * 4] * TIME_TICK;
//        position_and_weight[idx * 4 + 1] += m_deviceVelocity[idx * 4 + 1] * TIME_TICK;
//        position_and_weight[idx * 4 + 2] += m_deviceVelocity[idx * 4 + 2] * TIME_TICK;
//
//        // Recompute acceleration for the second half-step velocity update
//        accelerate = { 0.0f, 0.0f, 0.0f };
//        now_pos = 0;
//
//        // Synchronize threads to ensure all threads have updated positions
//        __syncthreads();
//
//        main_body[0] = position_and_weight[idx * 4];
//        main_body[1] = position_and_weight[idx * 4 + 1];
//        main_body[2] = position_and_weight[idx * 4 + 2];
//        main_body[3] = position_and_weight[idx * 4 + 3];
//
//        while (now_pos < 4 * N) {
//            mass[k * 4] = position_and_weight[now_pos + k * 4];
//            mass[k * 4 + 1] = position_and_weight[now_pos + k * 4 + 1];
//            mass[k * 4 + 2] = position_and_weight[now_pos + k * 4 + 2];
//            mass[k * 4 + 3] = position_and_weight[now_pos + k * 4 + 3];
//
//            now_pos += BLOCK_SIZE * 4;
//            __syncthreads();
//
//            for (int i = 0; i < BLOCK_SIZE; i++) {
//                accelerate = cal_single_acclerate(main_body, &mass[i * 4], accelerate);
//            }
//            __syncthreads();
//        }
//
//        // Update velocity with second half-step acceleration
//        m_deviceVelocity[idx * 4] += accelerate.x * (TIME_TICK / 2.0f);
//        m_deviceVelocity[idx * 4 + 1] += accelerate.y * (TIME_TICK / 2.0f);
//        m_deviceVelocity[idx * 4 + 2] += accelerate.z * (TIME_TICK / 2.0f);
//    }
//}

/* #endregion --------------------简单，但确实有效的方法-------------------- */



/* #region --------------------串行计算方法--------------------*/

// 单线程串行版本的核函数
__global__ void single_thread_update_all(real* m_deviceVelocity, real* position_and_weight, int N) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int idx = 0; idx < N; idx++) {
            real main_body[4];
            main_body[0] = position_and_weight[idx * 4];
            main_body[1] = position_and_weight[idx * 4 + 1];
            main_body[2] = position_and_weight[idx * 4 + 2];
            main_body[3] = position_and_weight[idx * 4 + 3];
            float3 accelerate = { 0.0f, 0.0f, 0.0f };

            for (int j = 0; j < N; j++) {
                if (idx != j) {
                    real other_body[4];
                    other_body[0] = position_and_weight[j * 4];
                    other_body[1] = position_and_weight[j * 4 + 1];
                    other_body[2] = position_and_weight[j * 4 + 2];
                    other_body[3] = position_and_weight[j * 4 + 3];
                    accelerate = cal_single_acclerate(main_body, other_body, accelerate);
                }
            }

            // 更新速度
            m_deviceVelocity[idx * 4] += accelerate.x * TIME_TICK;
            m_deviceVelocity[idx * 4 + 1] += accelerate.y * TIME_TICK;
            m_deviceVelocity[idx * 4 + 2] += accelerate.z * TIME_TICK;

            // 更新位置
            position_and_weight[idx * 4] += m_deviceVelocity[idx * 4] * TIME_TICK;
            position_and_weight[idx * 4 + 1] += m_deviceVelocity[idx * 4 + 1] * TIME_TICK;
            position_and_weight[idx * 4 + 2] += m_deviceVelocity[idx * 4 + 2] * TIME_TICK;
        }
    }
}

/* #endregion --------------------串行计算方法--------------------*/


__global__ void printFirstDataPoint(real* data) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {  // 确保只有一个线程执行打印
        for (int i = 0;i < 5;i++) {
            printf("No.%d data point: x=%f, y=%f, z=%f, w=%f\n", i, data[0+4*i], data[1+4*i], data[2+4*i], data[3+4*i]);
        }
    }
}

__global__ void findMaxAbsValues(real* data, int numPoints, real* maxValues) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {  // 确保只有一个线程执行
        real maxX = 0;
        real maxY = 0;
        real maxZ = 0;
        real maxW = 0;

        // 遍历所有点
        for (int i = 0; i < numPoints; i++) {
            real x = fabs(data[4 * i]);     // x坐标的绝对值
            real y = fabs(data[4 * i + 1]); // y坐标的绝对值
            real z = fabs(data[4 * i + 2]); // z坐标的绝对值
            real w = fabs(data[4 * i + 3]); // w坐标的绝对值

            // 更新最大值
            if (x > maxX) maxX = x;
            if (y > maxY) maxY = y;
            if (z > maxZ) maxZ = z;
            if (w > maxW) maxW = w;
        }

        // 将结果存储在maxValues数组中
        maxValues[0] = maxX;
        maxValues[1] = maxY;
        maxValues[2] = maxZ;
        maxValues[3] = maxW;
    }
}

//__global__ void


void resizeWindowCallback(GLFWwindow* window, int width, int height)
{
    // 更新 OpenGL 视口大小
    glViewport(0, 0, width, height);
}

void load_data(int choice) {
    if (choice == 0) { // 一个简单的旋转星系
        loadTipsyFile("./data/galaxy_20K.bin");
        scaleFactors[0] = 200;
        scaleFactors[1] = 120;
        scaleFactors[2] = 200;
    }
    else if (choice == 1) { // 两个旋转星系相撞
        loadTabFile("./data/dubinski.tab");
        scaleFactors[0] = 100;
        scaleFactors[1] = 100;
        scaleFactors[2] = 100;
    }
    else if (choice == 2) { // 粒子数量众多的一个简单中心星系
        loadTabFile("./data/tab65536.tab");
        scaleFactors[0] = 50;
        scaleFactors[1] = 50;
        scaleFactors[2] = 50;
    }
    else if (choice == 3) { // 星团爆炸
        loadDatFile("./data/stars.dat");
        scaleFactors[0] = 400;
        scaleFactors[1] = 400;
        scaleFactors[2] = 400;
        cameraDistance = 3;
    }
    else if (choice == 4) { // 十帧电竞
        loadDatFile("./data/k17c.snap");
        scaleFactors[0] = 300;
        scaleFactors[1] = 300;
        scaleFactors[2] = 300;
    }
    else if (choice == 5) {
        loadDatFile("./data/k17hp.snap");
        scaleFactors[0] = 300;
        scaleFactors[1] = 300;
        scaleFactors[2] = 300;
    }
}


void scroll_callback(GLFWwindow* window, double xoffset, double yoffset) {
    if (yoffset < 0) {
        cameraDistance *= 1.1f;
    }
    else if (yoffset > 0) {
        cameraDistance *= 0.9f;
    }
}

void mouse_button_callback(GLFWwindow* window, int button, int action, int mods) {
    if (button == GLFW_MOUSE_BUTTON_LEFT) {
        if (action == GLFW_PRESS) {
            isMousePressed = true;
            glfwGetCursorPos(window, &lastMouseX, &lastMouseY);
        }
        else if (action == GLFW_RELEASE) {
            isMousePressed = false;
        }
    }
}

void cursor_position_callback(GLFWwindow* window, double xpos, double ypos) {
    if (isMousePressed) {
        double deltaX = xpos - lastMouseX;
        double deltaY = lastMouseY - ypos;

        float sensitivity = 0.1f; // 鼠标移动的灵敏度

        sphereCoords.x -= deltaX * sensitivity;
        sphereCoords.y -= deltaY * sensitivity;

        // 限制phi的范围在-90到90度之间
        sphereCoords.y = glm::clamp(sphereCoords.y, -89.0f, 89.0f);

        lastMouseX = xpos;
        lastMouseY = ypos;
    }
}

void updateCameraPos() {
    float theta = glm::radians(sphereCoords.x);
    float phi = glm::radians(sphereCoords.y);

    float x = cos(phi) * sin(theta);
    float y = sin(phi);
    float z = cos(phi) * cos(theta);

    cameraPos = glm::vec3(x, y, z);
}




int main(int argc, char** argv) {

    int DATA_ID = 0;
    if (argc > 1) {
        try {
            DATA_ID = std::stoi(argv[1]); // 将第一个命令行参数转为整数并赋值给value
        }
        catch (const std::invalid_argument& e) {
            std::cerr << "Invalid argument: " << argv[1] << " is not a valid number." << std::endl;
            return 1;
        }
        catch (const std::out_of_range& e) {
            std::cerr << "Out of range: " << argv[1] << " is out of range for an int." << std::endl;
            return 1;
        }
    }
    if (DATA_ID > 5) {
        printf("DATA_ID should be 0-5.\n");
        exit(0);
    }

    // 初始化和设置窗口

    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    GLFWwindow* window = glfwCreateWindow(2400, 1800, "Point Sprites", nullptr, nullptr);
    if (!window) {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    
    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, resizeWindowCallback);
    // 初始化glad
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }


    // 设置键盘回调函数
    glfwSetMouseButtonCallback(window, mouse_button_callback);
    glfwSetCursorPosCallback(window, cursor_position_callback);

    // 设置和编译着色器
    GLuint shaderProgram = loadShader("vertex_shader.glsl", "fragment_shader.glsl");

    // 读入全部的顶点数据
    load_data(DATA_ID);
    // 临时更新点数量
    m_numBodies = 15361;

    printf("read star data done.\n");
    printf("num of Bodies = %d\n", m_numBodies);

    // 映射初始化好的buf资源
    hipGraphicsMapResources(1, &m_pGRes[0], 0); //第一个参数是映射的buf数量，第二个参数是资源的指针的数组， 第三个参数是流
    void* devPtr;
    size_t size;
    hipGraphicsResourceGetMappedPointer(&devPtr, &size, m_pGRes[0]);
    printf("initial_len: %d Bytes\n", size);
    real* devMaxValues;
    hipMalloc(&devMaxValues, 4 * sizeof(real));
    // 读入第一个点的xyzw数据并打印
    printFirstDataPoint <<<1, 1>>> ((real*)devPtr);
    hipDeviceSynchronize();

    // 调用内核
    findMaxAbsValues <<<1, 1 >>> ((real*)devPtr, 20000, devMaxValues);
    // 获取结果
    real maxValues[4];
    hipMemcpy(maxValues, devMaxValues, 4 * sizeof(real), hipMemcpyDeviceToHost);
    // 打印结果
    printf("Max absolute values: X=%f, Y=%f, Z=%f, W=%f\n", maxValues[0], maxValues[1], maxValues[2], maxValues[3]);
    hipGraphicsUnmapResources(1, &m_pGRes[0], 0); // 使用完毕后释放资源，交还给openGL
    printf(" return control to openGL\n");

    

    // 总任务数量是: ALL_TASK_CNT = body_num*(body_num-1)//2 = 20224*10112 = 204,505,088 个计算任务
    // 一共有128个SM
    // 每个SM负载1,597,696个计算任务
    // 每个Block最多1024个线程，从而平均每个线程1569.25个计算任务
    // 从而设置TASK_BLOCK_SIZE = 40
    // 一个任务块是: TASK_BLOCK_SIZE = *1024 = 1,048,576 个计算任务
    // 总共需要的任务块数量: BLOCK_CNT = ALL_TASK_CNT/TASK_BLOCK_SIZE = 195
    // 一个任务块执行所需内存加载量(Byte): 1024*2*size(float) = 8192 bytes
    // 一个SM的总共享内存量: 102400 bytes

    // 一个block的最大共享内存量: 49152 bytes
    // 每个任务块对应一个线程，对应需要加载1024 Bytes到共享内存
    int body_num = m_numBodies;
    int all_task_num = m_numBodies * (m_numBodies - 1) / 2;
    int task_block_edge_len = BLOCK_SIZE; // 任务块大小=block内线程数量
    int hori_group_nums = (body_num - 2) / task_block_edge_len + 1; // 多减了一个1是因为如果有N个物品，则对应一行只有N-1个任务，(0,0)不计算
    int block_num = hori_group_nums * (hori_group_nums + 1) / 2; // 计算引力矩阵的线程数量

    // 分配引力数组
    float3* gravity_array; // 各个天体之间的引力matrix拉伸成一个数组
    hipError_t status = hipMalloc((void**)&gravity_array, all_task_num * sizeof(float3));

    float3* gravity_sum_array;
    status = hipMalloc((void**)&gravity_sum_array, m_numBodies * sizeof(float3));

    if (status != hipSuccess) {
        printf("gravity_array CUDA malloc failed: %s\n", hipGetErrorString(status));
        return -1;
    }



    GLuint VAO;
    glGenVertexArrays(1, &VAO);
    glBindVertexArray(VAO);
    glBindBuffer(GL_ARRAY_BUFFER, m_pbo[m_currentRead]);

    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 4 * sizeof(real), (void*)0); // 位置属性
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, 4 * sizeof(real), (void*)(3 * sizeof(real))); // 质量属性，假设为点大小
    glEnableVertexAttribArray(1);

    // 解绑操作，确保不再有任何顶点缓冲区（VBO）和顶点数组对象（VAO）被当前的OpenGL上下文绑定
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glBindVertexArray(0);


    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE);

    /*
     * 有多个列表需要理清
     * 1. 两个位置+重量的缓冲区：m_pGRes[0]和m_pGRes[1],大小为m*4*size(real), 需要使用hipGraphicsMapResources函数来获取对应的地址指针
     *    计算引力的时候，需要将当前的读缓冲区的内容加载到共享内存里
     * 2. gravity_array， 引力数组大小为 m(m-1)/2 * size(real)
     */


    /* debug */


    // 调试用
    //hipGraphicsMapResources(1, &m_pGRes[0], 0);
    //size_t new_size;
    //hipGraphicsResourceGetMappedPointer(&position_and_weight, &new_size, m_pGRes[0]);
    //printf("new_resource_len: %d Bytes\n", new_size); // 327680 Bytes
    //hipGraphicsUnmapResources(1, &m_pGRes[0], 0); // 使用完毕后释放资源，交还给openGL

    void* position_and_weight;
    size_t resourse_size;
    int share_memory_size = BLOCK_SIZE * sizeof(real) * 4;//每个天体需要4个real类型长度,每个block存task_block_edge_len个点
    int count = 0;
    float frame_count = 0;
    //glfwSetCursorPosCallback(window, mouse_callback);
    //glfwSetMouseButtonCallback(window, mouse_button_callback);
    //glfwSetScrollCallback(window, scroll_callback);

    glfwSetScrollCallback(window, scroll_callback);
    
    GLint scaleFactorsLocation = glGetUniformLocation(shaderProgram, "scaleFactors");
    GLint viewLocation = glGetUniformLocation(shaderProgram, "view");
    glm::mat4 projection = glm::perspective(glm::radians(45.0f), (float)2400 / (float)1800, 0.1f, 100.0f);
    GLint projectionLocation = glGetUniformLocation(shaderProgram, "projection");

    // 渲染循环
    while (!glfwWindowShouldClose(window)) {
        processInput(window);
        // std::cout << "camera pos: (" << cameraPos[0] << "," << cameraPos[1] << "," << cameraPos[2] << ")\n" << std::endl;
        auto frame_start = std::chrono::high_resolution_clock::now();
        
        //glm::mat4 view = glm::lookAt(cameraPos, cameraPos + cameraFront, cameraUp);
        //glm::mat4 projection = glm::perspective(glm::radians(fov), 800.0f / 600.0f, 0.1f, 100.0f);

        //printMatrix(view, "View Matrix");
        //printMatrix(projection, "Projection Matrix");

        //// 传递矩阵到着色器
        //GLuint viewLoc = glGetUniformLocation(shaderProgram, "view");
        //GLuint projLoc = glGetUniformLocation(shaderProgram, "projection");
        //glUniformMatrix4fv(viewLoc, 1, GL_FALSE, glm::value_ptr(view));
        //glUniformMatrix4fv(projLoc, 1, GL_FALSE, glm::value_ptr(projection));

        
        glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT); // 清理缓冲区
        // printf("running %d\n", count);


        auto start_compute = std::chrono::high_resolution_clock::now();
        #if VERSION == 0
            // 简单模拟法
            hipGraphicsMapResources(1, &m_pGRes[0], 0);
            hipGraphicsResourceGetMappedPointer(&position_and_weight, &resourse_size, m_pGRes[0]);
            cal_gravity_new << <block_num, task_block_edge_len, share_memory_size >> > (gravity_array, (real*)position_and_weight, task_block_edge_len, hori_group_nums, m_numBodies);
            hipDeviceSynchronize();
            auto compute_point_1 = std::chrono::high_resolution_clock::now();
            add_up_gravity << < (m_numBodies - 1) / 256 + 1, 256 >> > (gravity_array, gravity_sum_array, m_numBodies);
            hipDeviceSynchronize();
            auto compute_point_2 = std::chrono::high_resolution_clock::now();
            update_position_and_speed << <(m_numBodies - 1) / 512 + 1, 512 >> > (m_deviceVelocity, gravity_sum_array, (real*)position_and_weight, m_numBodies);
            hipDeviceSynchronize();
            hipGraphicsUnmapResources(1, &m_pGRes[0], 0); // 使用完毕后释放资源，交还给openGL
        #elif VERSION == 1
            // 两步模拟法
            hipGraphicsMapResources(1, &m_pGRes[0], 0);
            hipGraphicsResourceGetMappedPointer(&position_and_weight, &resourse_size, m_pGRes[0]);
            cal_gravity << <block_num, task_block_edge_len, share_memory_size >> > (gravity_array, (real*)position_and_weight, task_block_edge_len, hori_group_nums, m_numBodies);
            hipDeviceSynchronize();
            add_up_gravity << < (m_numBodies - 1) / 1024 + 1, 1024 >> > (gravity_array, gravity_sum_array, m_numBodies);
            hipDeviceSynchronize();
            update_speed_half << <(m_numBodies - 1) / 1024 + 1, 1024 >> > (m_deviceVelocity, gravity_sum_array, (real*)position_and_weight, m_numBodies);
            hipDeviceSynchronize();
            update_position_complete << <(m_numBodies - 1) / 1024 + 1, 1024 >> > (m_deviceVelocity, gravity_sum_array, (real*)position_and_weight, m_numBodies);
            hipDeviceSynchronize();
            cal_gravity << <block_num, task_block_edge_len, share_memory_size >> > (gravity_array, (real*)position_and_weight, task_block_edge_len, hori_group_nums, m_numBodies);
            hipDeviceSynchronize();
            add_up_gravity << < (m_numBodies - 1) / 1024 + 1, 1024 >> > (gravity_array, gravity_sum_array, m_numBodies);
            hipDeviceSynchronize();
            update_speed_half << <(m_numBodies - 1) / 1024 + 1, 1024 >> > (m_deviceVelocity, gravity_sum_array, (real*)position_and_weight, m_numBodies);
            hipDeviceSynchronize();
            hipGraphicsUnmapResources(1, &m_pGRes[0], 0); // 使用完毕后释放资源，交还给openGL
        #elif VERSION == 2
            // 大道至简法
            hipGraphicsMapResources(1, &m_pGRes[0], 0);
            hipGraphicsResourceGetMappedPointer(&position_and_weight, &resourse_size, m_pGRes[0]);
            simple_update_all << < (m_numBodies - 2) / BLOCK_SIZE + 1, BLOCK_SIZE, share_memory_size >> > ((real*)m_deviceVelocity, (real*)position_and_weight, m_numBodies-1);
            hipDeviceSynchronize();
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                printf("simple_update_all: CUDA Error: %s\n", hipGetErrorString(err));
            }
            hipGraphicsUnmapResources(1, &m_pGRes[0], 0); // 使用完毕后释放资源，交还给openGL
        
        #elif VERSION == 3
            // 串行方法
            hipGraphicsMapResources(1, &m_pGRes[0], 0);
            hipGraphicsResourceGetMappedPointer(&position_and_weight, &resourse_size, m_pGRes[0]);
            single_thread_update_all <<< 1,1 >>> ((real*)m_deviceVelocity, (real*)position_and_weight, m_numBodies-1);
            hipDeviceSynchronize();
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                printf("simple_update_all: CUDA Error: %s\n", hipGetErrorString(err));
            }
            hipGraphicsUnmapResources(1, &m_pGRes[0], 0); // 使用完毕后释放资源，交还给openGL
        #elif VERSION == 4
            // 优化后的自定义方法
            hipGraphicsMapResources(1, &m_pGRes[0], 0);
            hipGraphicsResourceGetMappedPointer(&position_and_weight, &resourse_size, m_pGRes[0]);

            // 将所有的加速度计算好放在acc_array中（为了方便使用各种方法进行运行，这里使用了gravity_array这个变量名）
            cal_acc_new<<<block_num, task_block_edge_len, share_memory_size*2 >>> (gravity_array, (real*)position_and_weight, task_block_edge_len, hori_group_nums, m_numBodies); // 这里名为gravity_array，实为acc_array
            hipDeviceSynchronize();
            auto compute_point_1 = std::chrono::high_resolution_clock::now();
            // 使用计算好的acc来更新位置
            use_acc_update_position << <(m_numBodies - 1) / BLOCK_SIZE + 1, BLOCK_SIZE >> > ((real*)position_and_weight, (real*)m_deviceVelocity, gravity_array);
            hipDeviceSynchronize();
            auto compute_point_2 = std::chrono::high_resolution_clock::now();
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                printf("simple_update_all: CUDA Error: %s\n", hipGetErrorString(err));
            }
            hipGraphicsUnmapResources(1, &m_pGRes[0], 0); // 使用完毕后释放资源，交还给openGL
        #elif VERSION == 5
            // 第二次优化后的自定义方法
            hipGraphicsMapResources(1, &m_pGRes[0], 0);
            hipGraphicsResourceGetMappedPointer(&position_and_weight, &resourse_size, m_pGRes[0]);
            // 将所有的加速度计算好放在acc_array中（为了方便使用各种方法进行运行，这里使用了gravity_array这个变量名）
            cal_acc_advanced << <block_num, task_block_edge_len, share_memory_size * 2 >> > (gravity_array, (real*)position_and_weight, task_block_edge_len, hori_group_nums, m_numBodies); // 这里名为gravity_array，实为acc_array
            hipDeviceSynchronize();
            auto compute_point_1 = std::chrono::high_resolution_clock::now();
            // 使用计算好的acc来更新位置
            use_acc_update_position << <(m_numBodies - 1) / BLOCK_SIZE + 1, BLOCK_SIZE >> > ((real*)position_and_weight, (real*)m_deviceVelocity, gravity_array);
            hipDeviceSynchronize();
            auto compute_point_2 = std::chrono::high_resolution_clock::now();
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                printf("simple_update_all: CUDA Error: %s\n", hipGetErrorString(err));
            }
            hipGraphicsUnmapResources(1, &m_pGRes[0], 0); // 使用完毕后释放资源，交还给openGL
        #endif

        auto end_compute = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> compute_time = end_compute - start_compute;
        std::cout << "Total Compute Time:" << compute_time.count() << " ms" << std::endl;
        //std::chrono::duration<double, std::milli> compute_time_1 = compute_point_1 - start_compute;
        //std::cout << "start-point1 Time:" << compute_time_1.count() << " ms" << std::endl;
        //std::chrono::duration<double, std::milli> compute_time_2 = compute_point_2 - compute_point_1;
        //std::cout << "point1-point2 Time:" << compute_time_2.count() << " ms" << std::endl;
        //std::chrono::duration<double, std::milli> compute_time_3 = end_compute - compute_point_2;
        //std::cout << "point2-end Time:" << compute_time_3.count() << " ms" << std::endl;

        // 绘制点精灵
        auto start_render = std::chrono::high_resolution_clock::now();
        glUseProgram(shaderProgram);
        glUniform3fv(scaleFactorsLocation, 1, scaleFactors);
        // 传递当前视角矩阵到着色器
        updateCameraPos();
        glm::mat4 view = glm::lookAt(cameraPos * cameraDistance,   //摄像机的位置向量
                                     cameraTarget,   //目标位置向量
                                     cameraUp);  //上向量
        glUniformMatrix4fv(viewLocation, 1, GL_FALSE, glm::value_ptr(view));
        // 把投影矩阵传给着色器
        glUniformMatrix4fv(projectionLocation, 1, GL_FALSE, glm::value_ptr(projection));
        // 重新绑定VAO和VBO
        glBindVertexArray(VAO);
        glBindBuffer(GL_ARRAY_BUFFER, m_pbo[0]);
        glEnable(GL_PROGRAM_POINT_SIZE);
        glDrawArrays(GL_POINTS, 0, real_body_nums); // 只显示真实的点，填充的点不渲染
        glBindVertexArray(0);
        auto end_render = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> render_time = end_render - start_render;
        std::cout << "render time:" << render_time.count() << " ms" << std::endl;
        auto frame_end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> frame_time = frame_end - frame_start;
        double frame_time_ms = frame_time.count();
        double fps = 1000.0 / frame_time_ms;

        // 更新窗口标题
        count++;
        frame_count += frame_time_ms;
        if (count == 100) {
            
            float average_frame = frame_count / count;
            count = 0;
            frame_count = 0;
            std::stringstream ss;
            ss << "Frame Time: " << average_frame << " ms | FPS: " << fps;
            printf("Frame Time:  %d ms", frame_time_ms);
            glfwSetWindowTitle(window, ss.str().c_str());
        }
        glfwSwapBuffers(window);
        glfwPollEvents();
        printf("------------------------------\n");
       // std::this_thread::sleep_for(std::chrono::seconds(5));
    }

    // 清理资源
    glDeleteVertexArrays(1, &VAO);
    glDeleteProgram(shaderProgram);
    hipFree(m_deviceVelocity);
    hipFree(gravity_array);
    hipFree(gravity_sum_array);
    hipGraphicsUnregisterResource(m_pGRes[0]);
    hipGraphicsUnregisterResource(m_pGRes[1]);
    glDeleteBuffers(2, m_pbo);

    glfwTerminate();
    return 0;
}